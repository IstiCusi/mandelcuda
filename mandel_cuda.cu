#include <hip/hip_runtime.h>
__device__ float mapIterationsToWavelength(int iterations, int maxIter, float minWavelength, float maxWavelength) {
    float logIter = logf(iterations);
    float logMaxIter = logf(maxIter);
    float normalizedLogIter = (logIter - logf(1)) / (logMaxIter - logf(1));
    float expScale = exp(normalizedLogIter * log(maxWavelength / minWavelength));
    float wavelength = minWavelength * expScale;
    return wavelength;
}

__global__ void mandelbrotKernel(int *output, float *wavelengths, float xmin, float xmax, float ymin, float ymax, int width, int height, int maxIter, float minWavelength, float maxWavelength) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        float x0 = xmin + (float)idx / width * (xmax - xmin);
        float y0 = ymin + (float)idy / height * (ymax - ymin);
        float x = 0.0f, y = 0.0f;

        int iteration = 0;
        while (x*x + y*y <= 4 && iteration < maxIter) {
            float xtemp = x*x - y*y + x0;
            y = 2*x*y + y0;
            x = xtemp;
            iteration++;
        }
        output[idy * width + idx] = iteration;
        wavelengths[idy * width + idx] = mapIterationsToWavelength(iteration, maxIter, minWavelength, maxWavelength);
    }
}

extern "C" void computeMandelbrot(int *output         , float *wavelengths     ,
                                  float xmin          , float xmax             ,
                                  float ymin          , float ymax             ,
                                  int width           , int height             ,
                                  int maxIter         ,
                                  float minWavelength , float maxWavelength) {
    int *d_output;
    float *d_wavelengths;
    size_t size = width * height * sizeof(int);
    size_t sizeWavelengths = width * height * sizeof(float);
    hipMalloc(&d_output, size);
    hipMalloc(&d_wavelengths, sizeWavelengths);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

    mandelbrotKernel<<<dimGrid, dimBlock>>>(d_output, d_wavelengths, xmin, xmax, ymin, ymax, width, height, maxIter, minWavelength, maxWavelength);

    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost); // Blocks automatically
    hipMemcpy(wavelengths, d_wavelengths, sizeWavelengths, hipMemcpyDeviceToHost);
    hipFree(d_output);
    hipFree(d_wavelengths);
}

